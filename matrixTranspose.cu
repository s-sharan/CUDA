#include "hip/hip_runtime.h"
#include <random>
#include <chrono>
#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "dev_array.h"
#include <math.h>
#include <fstream>

using namespace std;
using namespace std::chrono;

__global__ void transposeCoalescedKernel(float *odata, const float *idata, int N)
{
  const int size=64;
  int rows=N/size;
  __shared__ float tile[size][size];


  int x = blockIdx.x * size + threadIdx.x;
  int y = blockIdx.y * size + threadIdx.y;
  int width = gridDim.x * size;

  for (int j = 0; j < size; j += rows)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * size + threadIdx.x;  // transpose block offset
  y = blockIdx.x * size + threadIdx.y;

  for (int j = 0; j < size; j += rows)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

void transposeCoalesced(float *odata, const float *idata, int N){
    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block. 
    // a maximum of 512 threads can be assigned to a block
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
        if (N*N > 512){
            threadsPerBlock.x = 512;
            threadsPerBlock.y = 512;
            blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
            blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
        }
    transposeCoalescedKernel<<<blocksPerGrid,threadsPerBlock>>>(odata,idata, N);
}


int main()
{
    // Perform matrix multiplication C = A*B
    // where A, B and C are NxN matrices
    int N;
    cout<<"Enter the size of the arrays N:";
    cin>>N;
    int SIZE = N*N;
    char ch;
    cout<<"Do you want to perform computation on CPU also (y/n)";
    cin>>ch;
    bool flag=false;
    if(ch=='y') flag=true;

    // Using Uniform Random number generator to initialize the arrays.
  	default_random_engine generator(12312);
  	uniform_real_distribution<> distribution(-10.0,10.0);

  	// Making use of steady clock to measure the amount of time taken to compute the product of matrices.
  	steady_clock::time_point gpu_start;
    steady_clock::time_point gpu_end;
    steady_clock::time_point cpu_start;
    steady_clock::time_point cpu_end;
    duration<double> gpu_time_span;
    duration<double> cpu_time_span;

    // Allocate memory on the host
    vector<float> h_A(SIZE);
    vector<float> h_B(SIZE);

   

    // Initialize matrices on the host
    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            h_A[i*N+j] = distribution(generator);
        }
    }

    // Allocate memory on the device
    dev_array<float> d_A(SIZE);
    dev_array<float> d_B(SIZE);
    d_A.set(&h_A[0], SIZE);



    gpu_start = steady_clock::now();
    transposeCoalesced(d_B.getData(), d_A.getData(),N);
    hipDeviceSynchronize();
    
    d_B.get(&h_B[0], SIZE);
    hipDeviceSynchronize();
    gpu_end = steady_clock::now();
    gpu_time_span = duration_cast<duration<double>>(gpu_end - gpu_start);

    cout<<"Time taken to compute the product on a GPU: "<<gpu_time_span.count()<<endl;
    double err = 0;
    float *cpu_B;
    if(flag){
	    cpu_B=new float[SIZE];

	    // Now do the matrix multiplication on the CPU
	    cpu_start = steady_clock::now();
	    for (int row=0; row<N; row++){
	        for (int col=0; col<N; col++){
	            cpu_B[row*N+col]=h_A[col*N+row];
	        }
	    }
	    cpu_end = steady_clock::now();
	    cpu_time_span = duration_cast<duration<double>>(cpu_end - cpu_start);
	
    
	    cout<<"Time taken to compute the product on a CPU: "<<cpu_time_span.count()<<endl;
	}

	// Writing the input matrices and output matrices into files
    std::ofstream matA("transpose/inputMatrix.txt"); 
    std::ofstream cpu("transpose/cpu.txt");
    std::ofstream gpu("transpose/gpu.txt");   

	for (int ROW=0; ROW < N; ROW++){
        for (int COL=0; COL < N; COL++){
    		matA<<h_A[ROW * N + COL]<"\t";
    		if(flag) cpu<<cpu_B[ROW * N + COL]<"\t";
            gpu<<h_B[ROW * N + COL]<<"\t";
        }
        if(flag) cpu<<endl;
        gpu<<endl;
        matA<<endl;
    }

    if(flag) cout << "Normalised Error: " << err/SIZE << endl;

    return 0;
}
